#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>       // helper functions for CUDA timing and initialization
#include <helper_functions.h>  // helper functions for timing, string parsing

#include "../include/muonDeviceWrapperGPU.cuh"
#include "../include/HoughKernels.cuh"

#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iomanip>

namespace TrigMuonModuleKernels{

  DeviceManager::DeviceManager() : nDevices(0)
  {
    checkCudaErrors(hipGetDeviceCount(&nDevices));
  }

  APE::HoughTransformDeviceContext*
  DeviceManager::createHTContext(const int& devId){

    checkCudaErrors(hipSetDevice(devId));
    APE::HoughTransformDeviceContext* p = new APE::HoughTransformDeviceContext;
    p->m_deviceId = devId;
    
    checkCudaErrors(hipMalloc((void **)&p->d_HTConfig, sizeof(HT_ALGO_CONFIGURATION)));
    checkCudaErrors(hipMalloc((void **)&p->d_HTData, sizeof(INPUT_HT_DATA)));
    checkCudaErrors(hipHostMalloc((void **)&p->h_HTConfig, sizeof(HT_ALGO_CONFIGURATION)));
    checkCudaErrors(hipHostMalloc((void **)&p->h_HTData, sizeof(INPUT_HT_DATA)));

    return p;

  }


  void DeviceManager::deleteHTContext(APE::HoughTransformDeviceContext* p){

    checkCudaErrors(hipSetDevice(p->m_deviceId));
    
    checkCudaErrors(hipFree(p->d_HTConfig));
    checkCudaErrors(hipFree(p->d_HTData));
    checkCudaErrors(hipHostFree(p->h_HTConfig));
    checkCudaErrors(hipHostFree(p->h_HTData));

    delete p;

  }

  float wrpHoughCtx(const APE::HoughTransformDeviceContext& devC){
    
    checkCudaErrors(hipSetDevice(devC.m_deviceId));
    //checkCudaErrors(hipMemcpy(devC.d_HTConfig, devC.h_HTConfig, sizeof(HT_ALGO_CONFIGURATION), hipMemcpyHostToDevice));

    double * curvGM;
    checkCudaErrors(hipMalloc((void**)&curvGM, sizeof(double) * curvBins));

    copyCurvVals<<<1, curvBins*0.5>>>(curvGM);
    checkCudaErrors(hipStreamSynchronize(0));

    copyCfgData((HT_ALGO_CONFIGURATION*) devC.h_HTConfig, curvGM);
    checkCudaErrors(hipStreamSynchronize(0));

    return 0.;

  };

  std::vector<float> wrpHoughAlgo(const APE::HoughTransformDeviceContext& devC, MUON_HOUGH_RED_PATTERN *pOutput){

    std::vector<float> timeVec;

    struct timeval tStart, tMid1, tMid2, tEnd;
    float totalCUDATime = 0.;

    gettimeofday (&tStart, NULL);

    HT_ALGO_CONFIGURATION * hConf = reinterpret_cast<HT_ALGO_CONFIGURATION*>(devC.h_HTConfig);
    INPUT_HT_DATA* hData = reinterpret_cast<INPUT_HT_DATA*>(devC.h_HTData);
    int Nsec[2] = {hConf->steps.sectors.xyz, hConf->steps.sectors.rz};
    int NA[2] = {(int)(2*hConf->steps.ip.xy/hConf->steps.stepsize.xy), (int)hConf->steps.nbins_curved};//+2
    //S:12,16 A:16,160 B:1440,720
    int voteXY = hData->m_nVoteXY;
    int pattXY = hData->m_nPattXY;
    int voteCC = hData->m_nVoteRZ;
    int pattCC = hData->m_nPattRZ;

    gettimeofday (&tMid1, NULL);
    std::cout << "TOTAL Host Preliminary Stuff " << (((tMid1.tv_sec - tStart.tv_sec)*1000000L +tMid1.tv_usec) - tStart.tv_usec) * 0.001 << " ms" <<  std::endl;

    // stomp a foot on the device    
    checkCudaErrors(hipSetDevice(devC.m_deviceId));

    hipDeviceProp_t prop;
    checkCudaErrors(hipGetDeviceProperties(&prop, devC.m_deviceId));
    int gpuProps[2] = {prop.warpSize, prop.maxThreadsPerBlock};

    gettimeofday (&tMid2, NULL);
    std::cout << "TOTAL DEVICE Preliminary Stuff " << (((tMid2.tv_sec - tMid1.tv_sec)*1000000L +tMid2.tv_usec) - tMid1.tv_usec) * 0.001 << " ms" <<  std::endl;

    // create and start CUDA timer
    StopWatchInterface *timerCUDA = 0;
    sdkCreateTimer(&timerCUDA);
    sdkResetTimer(&timerCUDA);
    sdkStartTimer(&timerCUDA);

    // Copy from Host to Device
    checkCudaErrors(hipMemcpyAsync(devC.d_HTData, devC.h_HTData, sizeof(INPUT_HT_DATA), hipMemcpyHostToDevice, 0));

    // allocate output mem area
    MUON_HOUGH_RED_PATTERN * pOut;
    checkCudaErrors(hipMalloc((void **)&pOut, sizeof(MUON_HOUGH_RED_PATTERN)));
    //checkCudaErrors(hipMemcpyAsync(pOut, pOutput, sizeof(MUON_HOUGH_RED_PATTERN), hipMemcpyHostToDevice, 0));

    int * votXYHit, * assXYHit, * votCCHit, * assCCHit;
    checkCudaErrors(hipMalloc((void **)&votXYHit, sizeof(int) * voteXY));
    checkCudaErrors(hipMalloc((void **)&assXYHit, sizeof(int) * pattXY));
    checkCudaErrors(hipMalloc((void **)&votCCHit, sizeof(int) * voteCC));
    checkCudaErrors(hipMalloc((void **)&assCCHit, sizeof(int) * pattCC));
    /*
    int * b_xy_maxes, * v_xy_maxes;
    checkCudaErrors(hipMalloc(&b_xy_maxes, sizeof(int) * Nsec[0] * NA[0]));
    checkCudaErrors(hipMalloc(&v_xy_maxes, sizeof(int) * Nsec[0] * NA[0]));*/
    int * b_cc_maxes, * v_cc_maxes;
    checkCudaErrors(hipMalloc(&b_cc_maxes, sizeof(int) * Nsec[1] * NA[1]));
    checkCudaErrors(hipMalloc(&v_cc_maxes, sizeof(int) * Nsec[1] * NA[1]));

    int * devProps;
    checkCudaErrors(hipMalloc((void**)&devProps, 2*sizeof(int)));
    checkCudaErrors(hipMemcpyAsync(devProps, gpuProps, 2*sizeof(int), hipMemcpyHostToDevice, 0));
    /*
    int * s_xy_max, * b_xy_max, * v_xy_max;
    checkCudaErrors(hipMalloc(&s_xy_max, sizeof(int) * Nsec[0]));
    checkCudaErrors(hipMalloc(&b_xy_max, sizeof(int) * Nsec[0]));
    checkCudaErrors(hipMalloc(&v_xy_max, sizeof(int) * Nsec[0]));*/
    int * s_cc_max, * b_cc_max, * v_cc_max;
    checkCudaErrors(hipMalloc(&s_cc_max, sizeof(int) * Nsec[1]));
    checkCudaErrors(hipMalloc(&b_cc_max, sizeof(int) * Nsec[1]));
    checkCudaErrors(hipMalloc(&v_cc_max, sizeof(int) * Nsec[1]));

    int * controls;
    checkCudaErrors(hipMalloc((void**)&controls, sizeof(int) * 4));

    sdkStopTimer(&timerCUDA);
    float TimerCUDASpan = sdkGetAverageTimerValue(&timerCUDA);

    timeVec.push_back(TimerCUDASpan);
    totalCUDATime += TimerCUDASpan;

    std::cout << "Input allocation time: " << TimerCUDASpan << " ms" << std::endl;

    sdkResetTimer(&timerCUDA);
    sdkStartTimer(&timerCUDA);

    int * monitor;
    checkCudaErrors(hipMalloc(&monitor, sizeof(int)*NA[1]));

    //houghAlgo<<< 1, 1 >>>(devProps, controls, votXYHit, assXYHit, votCCHit, assCCHit, (INPUT_HT_DATA*) devC.d_HTData, b_xy_maxes, v_xy_maxes, b_cc_maxes, v_cc_maxes, s_xy_max, b_xy_max, v_xy_max, s_cc_max, b_cc_max, v_cc_max, pOut);
    houghAlgo<<< 1, 1 >>>(devProps, controls, votXYHit, assXYHit, votCCHit, assCCHit, (INPUT_HT_DATA*) devC.d_HTData, b_cc_maxes, v_cc_maxes, s_cc_max, b_cc_max, v_cc_max, pOut, monitor);
    checkCudaErrors(hipStreamSynchronize(0));
    getLastCudaError("Wrapper Kernel execution failed");

    //int * h_xy=(int*)malloc(sizeof(int)*Nsec[0]*NA[0]);
    int * h_cc=(int*)malloc(sizeof(int)*Nsec[1]*NA[1]);
    //checkCudaErrors(hipMemcpyAsync((void*)h_xy, v_xy_maxes, sizeof(int)*Nsec[0]*NA[0], hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy((void*)h_cc, v_cc_maxes, sizeof(int)*Nsec[1]*NA[1], hipMemcpyDeviceToHost));
    /*
    std::cout << "XY\n";
    for(int i=0;i<Nsec[0]*NA[0];i++)
      std::cout << i << ":" << h_xy[i] << " ";*/
    std::cout << "\nCYLINDER\n";
    for(int i=0;i<Nsec[1]*NA[1];i++)
      std::cout << (int)((i+0.)/NA[1]) << ":" << h_cc[i] << " ";

    sdkStopTimer(&timerCUDA);
    TimerCUDASpan = sdkGetAverageTimerValue(&timerCUDA);


    int * h_mon=(int*)malloc(sizeof(int)*NA[1]);
    checkCudaErrors(hipMemcpy((void*)h_mon, monitor, sizeof(int)*NA[1], hipMemcpyDeviceToHost));
    std::cout << "\n";
    for(int y=0;y<NA[1];y++)
      std::cout << y << ":" << h_mon[y] << " ";//+0.)/100000. << " ";
    std::cout << "\n";

    timeVec.push_back(TimerCUDASpan);
    totalCUDATime += TimerCUDASpan;
    std::cout <<"Wrapper kernel execution " << TimerCUDASpan << " ms";// << std::endl;
 
    sdkResetTimer(&timerCUDA);
    sdkStartTimer(&timerCUDA);
    checkCudaErrors(hipMemcpyAsync((void*)pOutput, pOut, sizeof(MUON_HOUGH_RED_PATTERN), hipMemcpyDeviceToHost));
    sdkStopTimer(&timerCUDA);
    TimerCUDASpan = sdkGetAverageTimerValue(&timerCUDA);

    timeVec.push_back(TimerCUDASpan);
    totalCUDATime += TimerCUDASpan;
    std::cout <<"Output copy to host " << TimerCUDASpan << " ms" << std::endl;

    checkCudaErrors(hipFree(devProps));
    checkCudaErrors(hipFree(controls));/*
    checkCudaErrors(hipFree(b_xy_maxes));
    checkCudaErrors(hipFree(v_xy_maxes));*/
    checkCudaErrors(hipFree(b_cc_maxes));
    checkCudaErrors(hipFree(v_cc_maxes));/*
    checkCudaErrors(hipFree(s_xy_max));
    checkCudaErrors(hipFree(b_xy_max));
    checkCudaErrors(hipFree(v_xy_max));*/
    checkCudaErrors(hipFree(s_cc_max));
    checkCudaErrors(hipFree(b_cc_max));
    checkCudaErrors(hipFree(v_cc_max));
    checkCudaErrors(hipFree(votXYHit));
    checkCudaErrors(hipFree(assXYHit));
    checkCudaErrors(hipFree(votCCHit));
    checkCudaErrors(hipFree(assCCHit));
    checkCudaErrors(hipFree(pOut));

    checkCudaErrors(hipStreamSynchronize(0));
 
    gettimeofday (&tEnd, NULL);
    float totalRUNTime = (((tEnd.tv_sec - tStart.tv_sec)*1000000L +tEnd.tv_usec) - tStart.tv_usec) * 0.001;
    std::cout << "TOTAL RUNNING TIME " << totalRUNTime << " ms\nTOTAL CUDA TIME " << totalCUDATime << " ms" << std::endl;
    return timeVec;

  };

}